#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <time.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iterator>
#include <vector>

#define THREAD 256

#include <bits/stdc++.h>
#include <chrono>


using namespace std::chrono;

using namespace std;

__global__
void markPrimes(int step,int k, char *d_primes,  int n) {
	 int index = blockIdx.x*blockDim.x + threadIdx.x;
	
	 int begin = step * step;
	if (begin == 0){
		begin = step*step;
	}
	 int end = (index + 1) * k;
	for ( int i = begin; i <= end; i += step) //begin + step
	{
		if (i<n) 
			d_primes[i] = 0;
			
	}


}



int main(int argc, char *argv[]) {
	 int n, N, k, blocks;
	 char *primes, *d_primes;

	n = atoi(argv[1]); 
	N = ceill((long double)sqrt(n)); 
	k = ceill((long double)n / (long double)N); 
	blocks = ceill((long double)N / (long double)THREAD);

	primes = ( char*)malloc(n*sizeof(char));
	for (int i = 0; i < n; i++)
	{
			
			if (i%2!=0 || i==2 )
		    primes[i] = 1;
	}

	hipMalloc(&d_primes, n * sizeof(char));
	hipMemcpy(d_primes, primes, n * sizeof(char), hipMemcpyHostToDevice);
	clock_t begin = clock();
	for (int i = 2; i <= N; i++)
	{
		if (primes[i])
			{
			    markPrimes << <blocks, THREAD >> > (i, k, d_primes, n);
			    }
		
		
	}

	
		hipMemcpy(primes, d_primes, n * sizeof(char), hipMemcpyDeviceToHost);

		clock_t end = clock();

	
	primes[0] = 0;
	primes[1] = 0;
	 int count = 0;
	for (int i = 0; i < n; i++)
	{
		if (primes[i])
		{
			//cout << i << "\t";
			count++;
		}
	}
	cout << "\nNumber of primes less than " << n << ": " << count << endl;
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	cout << "The running time is " << time_spent << " milliseconds." << endl;


	hipFree(d_primes);
}